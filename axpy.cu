#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "race_detection.cuh"

__global__ void axpy(float a, float* x, float* y) {
//  int * _z = new int[16];
//  _z[ threadIdx.x] = x[threadIdx.x];
  y[threadIdx.x] = a * x[threadIdx.x];	
  __syncthreads();
//  delete[] _z;
}

int main(int argc, char* argv[]) {
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen];

  // Copy input data to device.
  float* device_x;
  float* device_y;
   checkCudaErrors(hipMalloc(&device_x, kDataLen * sizeof(float)));
   checkCudaErrors(hipMalloc(&device_y, kDataLen * sizeof(float)));
   checkCudaErrors(hipMemcpy(device_x, host_x, kDataLen * sizeof(float),
             hipMemcpyHostToDevice));

  // Launch the kernel.
  allocateReadWriteSets(1, kDataLen );
  axpy<<<1, kDataLen>>>(a, device_x, device_y);
  freeReadWriteSets( 1, kDataLen );

  // Copy output data to host.
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipMemcpy(host_y, device_y, kDataLen * sizeof(float),
             hipMemcpyDeviceToHost));

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << "\n";
  }

//  hipDeviceReset();
  return 0;
}
