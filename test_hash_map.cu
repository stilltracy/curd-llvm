#include "hip/hip_runtime.h"
#include "hash_map.cuh"

using namespace util;

__global__ void testHashMap( ConcurrentHashMap<int, int> * _hm, size_t nBuckets){
  for( auto i = 0; i < 32; i ++ ){
    _hm -> put( i, i * i ); 
  }
  for( auto i = 0; i < 32 ; i++ ){
    auto _v = _hm -> get( i );
    assert( *_v == i * i );
    printf("map[%d] : %d\n", i, *_v );
  }
  
}

int main(){
  size_t nBuckets = 16;
  typedef ConcurrentHashMap<int,int> map_type;
  auto _map = static_cast<map_type*>(map_type::makeDeviceHashMap( nBuckets ));
  testHashMap<<<4, 1 >>>( _map, nBuckets );
  checkCudaErrors( hipDeviceSynchronize() );
  
}
