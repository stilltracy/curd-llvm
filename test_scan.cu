#include "hip/hip_runtime.h"
#include "prefix_sum.cuh"
#include "utils.cuh"
#include <hip/hip_runtime_api.h>
using util::scan;
using util::printArray;
using util::ceil_div;

__global__ void testKernel(int * array, int size) {
  auto tid = threadIdx.x;
  auto nthreads = blockDim.x * blockDim.y * blockDim.z;
  auto iterations = ceil_div(size, nthreads);
  for (int i = 0; i < iterations; i++) {
    array[i * nthreads + tid] = tid % 2;
  }
  __syncthreads();
  printArray("before", array, size, tid);
  int sum = scan(tid, array, size, nthreads);
  if (!tid) {
    printf("sum: %d\n", sum);
  } 
  printArray("after", array, size, tid);
}

int main() {
  int * array;
  int nthreads = 512;
  int size = 2048;
  checkCudaErrors(hipMalloc(&array, sizeof(int) * size));
  testKernel<<<1, nthreads>>>(array, size);
  checkCudaErrors(hipDeviceSynchronize());
}
