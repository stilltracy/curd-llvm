#include "hip/hip_runtime.h"
#include "bitmap.cuh"
#include <hip/hip_runtime_api.h>

using util::Bitmap;

__global__ void testKernel() {
  const int size = 242;
  Bitmap<2> * _bitmap = new Bitmap<2>(size);
  char symbols[5] = "duxs";
  for (int i = 0; i < size; i++) {
    _bitmap -> set(i, i % 4);
  }
  _bitmap -> print(symbols, 64);
}

int main() {
  testKernel<<<1,1>>>();
  checkCudaErrors(hipDeviceSynchronize());
}
