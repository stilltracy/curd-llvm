#include "hip/hip_runtime.h"
#include "build_switches.cuh"
#include "race_detection.cuh"
#include "simple_set.cuh"
#include "lock.cuh"
#include "counters.cuh"

#include <inttypes.h>
#include <iostream>
#include <helper_string.h>    // helper for string parsing
#include <helper_image.h>     // helper for image and data comparison
#include <hip/hip_runtime_api.h>      // helper for cuda error checking functions

#include "debug_utils.cuh"
#include "curd_memory.cuh"
#include "memory_region.cuh"
#include "coalesced_sets.cuh"
#include "prefix_sum.cuh"
#include "race_buffer.cuh"
#include "config.h"
#include "utils.cuh"

using std::string;
using util::binary_search;
using util::check_order;
using util::Bitmap;

namespace curd{

typedef GlobalMemorySizeType Timestamp;
typedef SimpleArray<Timestamp> VectorClock;
typedef Simple2DArray<Timestamp> VectorClock2D;

typedef util::ConcurrentHashMap<uint64_t, CapabilitySet *, true > AtomicCSMap;
typedef util::ConcurrentHashMap<int, CapabilitySet *, true> BlockCSMap;
// here the key is the globally-linearized tid
typedef util::ConcurrentHashMap<uint64_t, CapabilitySet *, true> ThreadCSMap;
//typedef util::ConcurrentHashMap<int, ThreadCSMap *, true> BlockThreadCSMap;
typedef util::ConcurrentHashMap<uint64_t, TSPartitionsBuffer *, true> TSPBMap;

__constant__ size_t sharedMemorySizePerBlock;
__constant__ size_t totalGlobalMemory;


DEFINE_MAPPED_MEMORY_HANDLES( __constant__, CoalescedSetsShmem, shmemReadSets )
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, CoalescedSetsShmem, shmemWriteSets )
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, CoalescedSetsGlmem, glmemReadSets )
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, CoalescedSetsGlmem, glmemWriteSets )

DEFINE_MAPPED_MEMORY_HANDLES( __constant__, BlockSets, blockReadSets )
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, BlockSets, blockWriteSets )

DEFINE_MAPPED_MEMORY_HANDLES( __constant__, LocksDirectory, blockReadSetsLocksDir );
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, LocksDirectory, blockWriteSetsLocksDir );
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, VectorClock2D, blockVectorClocksDirectory );

__device__ GlobalMemorySizeType * _gmSizesBuffer;
GlobalMemorySizeType * _host_gmSizesBuffer;

DEFINE_MAPPED_MEMORY_HANDLES( __device__, int *, scanBuffer);
DEFINE_MAPPED_MEMORY_HANDLES( __device__, Bitmap<4>::HolderType *, bitmapBuffer);


__device__ Counters_shmemRd shmemRds;
__device__ Counters_shmemWr shmemWrs;
__device__ Counters_glmemRd glmemRds;
__device__ Counters_glmemWr glmemWrs;
__device__ Counters_barriers barriers;
__device__ Counters_interGWWRaces interGWWRaces;
__device__ Counters_interGRWRaces interGRWRaces;
__device__ Counters_interGWRRaces interGWRRaces;
__device__ Counters_smTSMerges smTSMerges;
__device__ Counters_gmTSMerges gmTSMerges;
__device__ Counters_brsWorstCount brsWorstCount;
__device__ Counters_bwsWorstCount bwsWorstCount;
__device__ Counters_brsActualCount brsActualCount;
__device__ Counters_bwsActualCount bwsActualCount;
__device__ Counters_maxSMTRSSize maxSMTRSSize;
__device__ Counters_maxGMTRSSize maxGMTRSSize;
__device__ Counters_maxBRSSize maxBRSSize;
__device__ Counters_maxSMTWSSize maxSMTWSSize;
__device__ Counters_maxGMTWSSize maxGMTWSSize;
__device__ Counters_maxBWSSize maxBWSSize;
__device__ Counters_intraBlockRaces intraBlockRaces;
__device__ Counters_entriesRemovedTRS entriesRemovedTRS;
__device__ Counters_initCycles initCycles;
__device__ Counters_SMChecksCycles SMChecksCycles;
__device__ Counters_dedupCycles dedupCycles;
__device__ Counters_BRSCompressCycles BRSCompressCycles;
__device__ Counters_BWSCompressCycles BWSCompressCycles;
__device__ Counters_GMIntraBChecksCycles GMIntraBChecksCycles;
__device__ Counters_GMCommitBufferCycles GMCommitBufferCycles;
__device__ Counters_sortTWSCycles sortTWSCycles;
__device__ Counters_sortTRSCycles sortTRSCycles;
__device__ Counters_intraBWWChecksCycles intraBWWChecksCycles;
__device__ Counters_intraBRWChecksCycles intraBRWChecksCycles;
__device__ Counters_ascendingInTWSS ascendingInTWSS;
__device__ Counters_descendingInTWSS descendingInTWSS;
__device__ Counters_ascendingInTS ascendingInTS;
__device__ Counters_descendingInTS descendingInTS;
__device__ Counters_maxReadSections maxReadSections;
__device__ Counters_maxWriteSections maxWriteSections;
__device__ Counters_allInOrder allInOrder;
__device__ Counters_allChecks allChecks;
__device__ Counters_irrelevantAccesses irrelevantAccesses;


DEFINE_MAPPED_MEMORY_HANDLES( __device__, RaceBuffer, racesDetected );
DEFINE_MAPPED_MEMORY_HANDLES( __constant__, Config, configParams );

size_t lastNThreads = 0;

__device__ inline SharedMemoryAddr toShmemAddr(uint64_t uintValue) {
  // TODO: what validation do we need here?
  //CURD_ASSERT_lt(uintValue, sharedMemorySizePerBlock, "shmem-uint", "SharedMemoryPerBlock");
  return (SharedMemoryAddr)uintValue;
}


#ifdef EAGER_INTERBLOCK_CHECK
//TODO: cannot be enabled together with eager-column-merge
//check for potential WAR races that would otherwise be missed silently (oldR vs. deltaRemoteW)
static __device__ inline size_t checkInterBlockWARs(
  int blockId, int localTid, int nthreads, int blocksNum,
  GlobalMemorySizeType oldRsSize, BlockSet * _localReadSet,
  GlobalMemorySizeType * remoteBlockWriteSetSizes ) {
  int iterations = ceil_div( blocksNum, nthreads );
  for(int j = 0; j < iterations; j++) {
    int remoteBlockId = j * nthreads + localTid;
    if (remoteBlockId < blocksNum && remoteBlockId != blockId) {
      Timestamp * _syncTime = blockVectorClocksDirectory.get(blockId, remoteBlockId);
      Timestamp lastSyncTime = *_syncTime;
      BlockSet * _remoteWriteSet = blockWriteSets.get( remoteBlockId );
      GlobalMemorySizeType remoteSize = remoteBlockWriteSetSizes[remoteBlockId];
      CURD_ASSERT_lt(lastSyncTime, remoteSize+1, "lastSync", "remoteSize+1");
      for( GlobalMemorySizeType i = 0 ; i < oldRsSize; i ++) {
        GlobalMemoryRegion & region = * _localReadSet -> get( i);
        int conflictsDetected = 0;
        for(Timestamp k = lastSyncTime ; k < remoteSize; k++) {
          auto _r = _remoteWriteSet -> get(k ) ;
          if( _r -> overlapsWith(region ) ){
            GlobalMemoryRegion intersection;
            intersection = _r -> intersect( region ) ;
            #ifdef WITH_SOURCE_INFO
            reportRace( conflictsDetected, RaceKind::InterBlock, RaceType::WR,
                                remoteBlockId, blockId, region.srcLoc, _r -> srcLoc, intersection );
            #else
            reportRace( conflictsDetected, RaceKind::InterBlock, RaceType::WR,
                                remoteBlockId, blockId,  intersection );

            #endif
           
            break;
          }
        }
      }
      *_syncTime = remoteBlockWriteSetSizes[remoteBlockId];
    }

  }
  return 0;
}
#endif

/* check intra-block races in a memory space ( shared / global )
  [write|read]SetsDirectory store pointers to the shadow memory
*/
template<typename RegionType, typename SizeType>
static __device__ inline size_t checkIntraBlockRaces(
  RaceKind kind,
  const CoalescedSets<RegionType, SizeType> & writeSetsDir,
  const CoalescedSets<RegionType, SizeType> & readSetsDir,
  int blockId, int localTid, int nthreads ) {
  using namespace util;
  // before doing inter-thread checks, each thread 
  // sorts its own set first
  // TODO: more efficient sorting here
  #ifdef ENABLE_SAMPLE_TIME
  auto lastClock = clock64();
  auto curClock = lastClock;
  #endif
  auto _setHead = writeSetsDir.getSetHead( blockId, localTid );
  auto wSetSize = *writeSetsDir.getPtrToSize( blockId, localTid );

  #ifdef BINARY_SEARCH
  int nAscending = 0;
  int allInOrderCount = 0;
  check_order(_setHead, wSetSize * nthreads, nthreads, nAscending);
  if (nAscending <= 1 || nAscending + 1>= wSetSize) {
    allInOrderCount++;
  } else {
    #ifdef SORT_TS
    bubble_sort( _setHead, wSetSize* nthreads, nthreads );
    #endif
  }
  ADD_COUNTER(ascendingInTS, nAscending);
  ADD_COUNTER(descendingInTS, wSetSize - nAscending);
  nAscending = 0;
  #endif
 
  SAMPLE_TIME(sortTWSCycles, curClock, lastClock); 

  _setHead = readSetsDir.getSetHead( blockId, localTid );
  auto rSetSize = *readSetsDir.getPtrToSize( blockId, localTid ); 
  #ifdef BINARY_SEARCH
  check_order(_setHead, rSetSize * nthreads, nthreads, nAscending);
  if (nAscending <= 1 || nAscending + 1>= rSetSize) {
    allInOrderCount++;
  } else {
    #ifdef SORT_TS
    bubble_sort( _setHead, rSetSize* nthreads, nthreads );
    #endif
  }
  ADD_COUNTER(ascendingInTS, nAscending);
  ADD_COUNTER(descendingInTS, rSetSize - nAscending);
  ADD_COUNTER(allInOrder, allInOrderCount);
  ADD_COUNTER(allChecks, 2);
  #endif
  SAMPLE_TIME( sortTRSCycles, curClock, lastClock); 
  __syncthreads();
  //  each thread reads WriteSets of all other threads
  //  (within the same block),  to check for conflicts ( WW/ RW)
  size_t nraces = 0;
  #ifdef INTRABLOCK_CHECKOVERLAPS
  nraces += writeSetsDir.checkIntraBlockOverlaps( kind, RaceType::WW, blockId, localTid );
  SAMPLE_TIME(intraBWWChecksCycles, curClock, lastClock); 
  if( wSetSize > rSetSize ){
    nraces += readSetsDir.checkIntraBlockOverlaps( kind, RaceType::RW, blockId, localTid, writeSetsDir ); 
  }else{
    nraces += writeSetsDir.checkIntraBlockOverlaps( kind, RaceType::WR, blockId, localTid, readSetsDir );
  }
  SAMPLE_TIME(intraBRWChecksCycles, curClock, lastClock); 
  #endif
  return nraces;
}

#ifdef EAGER_INTERBLOCK_CHECK
static __device__ inline void eagerInterBlockCheck(int oldWsSize, int oldRsSize,
    int deltaWsSize, int deltaRsSize ) {
  using namespace util;

  if( oldRsSize + deltaWsSize + deltaRsSize == 0) {
    return ;
  }
  int blockId = linearizedBlockId();
  if( blockId >= configParams.blocksTrackedGM ) {
    return;
  }
  int localTid= linearizedThreadId();
  int nthreads = blockDim.x * blockDim.y * blockDim.z;
  //int blocksNum = gridDim.x * gridDim.y * gridDim.z;
  int blocksNum = configParams.blocksTrackedGM;



  // we need to ensure that the sizes of all remote BlockWriteSets don't change
  // during these checks
  GlobalMemorySizeType * blockWriteSetsSizes =
    newArrayForBlock<GlobalMemorySizeType>( localTid, blocksNum);
  blockWriteSets.loadAllBlockSetSizes( blockId, localTid,
                                       blockWriteSetsLocksDir, blockWriteSetsSizes );
  // barrier shouldn't be necessary here, as there's a barrier in unlockForBlock
  //TODO: repetitive checks may happen here
  // delta( local Ws) vs. remote Ws
  __shared__ CapabilitySet * _bcs;
  if( localTid == 0 ){
    _bcs = getBlockCapabilitySet( blockId );  
  }
  __syncthreads();

  size_t WWChecks = blockWriteSets.checkInterBlockOverlaps(RaceType::WW, blockId, localTid,
                    nthreads, oldWsSize, deltaWsSize, blockWriteSetsSizes );
  // delta( local Rs) vs. remote Ws
  size_t RWChecks = blockReadSets.checkInterBlockOverlaps(RaceType::RW, blockId, localTid,
                    nthreads, oldRsSize, deltaRsSize, blockWriteSets, blockWriteSetsSizes);

  /*  barriers.print();
    interGWWRaces.print();
    interGRWRaces.print();
    printf("b%dt%d oldRsSize = %d, deltaRs = %d, oldWsSize= %d, deltaWsSize = %d\n",
      blockId, localTid, oldRsSize, deltaRsSize, oldWsSize, deltaWsSize );*/
  ADD_COUNTER( interGWWRaces, WWChecks );
  ADD_COUNTER( interGRWRaces, RWChecks );

  // delta( remote Ws') vs. local Rs
  //TODO: refactor this
  size_t WRChecks = checkInterBlockWARs( blockId, localTid, nthreads, blocksNum,
                                         oldRsSize, blockReadSets.get( blockId), blockWriteSetsSizes );
  ADD_COUNTER( interGWRRaces, WRChecks );

  deleteArrayForBlock( localTid, blockWriteSetsSizes );
}
#endif

}//end namespace curd

extern "C" {



  __device__ __inline__  void __curd__shmemReadHook(uint64_t uintValue, uint8_t width,
      uint32_t lineNo, char * filename) {
    using namespace curd;
    using namespace util;
    /*CURD_ASSERT( !isLocalPtr( uintValue) );
    CURD_ASSERT( !isConstantPtr( uintValue) );
    CURD_ASSERT( width );*/
#ifdef MONITOR_SHARED_MEMORY
    /*  printf("shmem read %ld, %d, %d, %p\n", uintValue, (uint32_t)width,
        lineNo, filename);*/
    INC_COUNTER( shmemRds );
    if( linearizedBlockId() >= configParams.blocksTrackedSM ) {
      return;
    }
    if(sharedMemorySizePerBlock == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }
    SharedMemoryAddr shmemAddr = toShmemAddr( uintValue );
    auto merged = shmemReadSets.add(SharedMemoryRegion(shmemAddr, width
#ifdef WITH_SOURCE_INFO
                                    , filename, lineNo
#endif
                                                      ))/*shmemAddr*/;//to prevent being optimized out 
    //TODO: ensure the readSets & writeSets are disjoint to improve performance
    ADD_COUNTER( smTSMerges, merged );
#endif
  }

  __device__ __inline__  void __curd__shmemWriteHook(uint64_t uintValue, uint8_t width,
      uint32_t lineNo, char * filename) {
    using namespace curd;
    using namespace util;
/*CURD_ASSERT( !isLocalPtr( uintValue) );
    CURD_ASSERT( !isConstantPtr( uintValue) );*/
    CURD_ASSERT( width);
#ifdef MONITOR_SHARED_MEMORY
    /*  printf("shmem write %ld, %d, %d, %p\n", uintValue, (uint32_t)width,
        lineNo, filename);*/
    INC_COUNTER( shmemWrs );
    if( linearizedBlockId() >= configParams.blocksTrackedSM ) {
      return;
    }
    if(sharedMemorySizePerBlock == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }
    SharedMemoryAddr shmemAddr = toShmemAddr( uintValue );
    auto merged = shmemWriteSets.add(SharedMemoryRegion(shmemAddr, width
#ifdef WITH_SOURCE_INFO
                                     , filename, lineNo
#endif
                                                       ))/*shmemAddr*/;//to prevent being optimized out 

    ADD_COUNTER( smTSMerges, merged );
#endif
  }

  __device__ __inline__ void __curd__glmemReadHook(uint64_t uintValue, uint8_t width,
      uint32_t lineNo, char * filename) {
    using namespace curd;
    using namespace util;
    /*CURD_ASSERT( !isLocalPtr( uintValue) );
    CURD_ASSERT( !isConstantPtr( uintValue) );*/
    CURD_ASSERT( width);
#ifdef MONITOR_GLOBAL_MEMORY
    /*  printf("glmem read %ld, %d, %d, %p\n", uintValue, (uint32_t)width,
        lineNo, filename);*/
    INC_COUNTER( glmemRds );
    if( linearizedBlockId() >= configParams.blocksTrackedGM ) {
      return;
    }
    if(totalGlobalMemory == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }

    GlobalMemoryAddr glmemAddr =static_cast<GlobalMemoryAddr>( uintValue);
    auto merged =glmemReadSets.add(GlobalMemoryRegion(glmemAddr, width
#ifdef WITH_SOURCE_INFO
                                    , filename, lineNo
#endif
                                                      ))/*glmemAddr*/;//to prevent being optimized out ;

    ADD_COUNTER( gmTSMerges, merged );
#endif
  }

  __device__ __inline__  void __curd__glmemWriteHook(uint64_t uintValue, uint8_t width,
      uint32_t lineNo, char * filename) {
    using namespace curd;
    using namespace util;
    /*CURD_ASSERT( !isLocalPtr( uintValue) );
    CURD_ASSERT( !isConstantPtr( uintValue) );*/
    CURD_ASSERT( width);
#ifdef MONITOR_GLOBAL_MEMORY
    /*  printf("glmem write %ld, %d, %d, %p\n", uintValue, (uint32_t)width,
        lineNo, filename);*/

    INC_COUNTER( glmemWrs );
    if( linearizedBlockId() >= configParams.blocksTrackedGM ) {
      return;
    }
    if(totalGlobalMemory == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }
    GlobalMemoryAddr glmemAddr =static_cast<GlobalMemoryAddr>( uintValue);
    auto merged = glmemWriteSets.add(GlobalMemoryRegion(glmemAddr, width
#ifdef WITH_SOURCE_INFO
                                     , filename, lineNo
#endif
                                                       ))/*glmemAddr*/;//to prevent being optimized out ;*/

    ADD_COUNTER( gmTSMerges, merged );
#endif
  }

  __device__ __inline__  void __curd__genericMemReadHook(uint64_t addr, uint8_t width, uint32_t lineNo, char * filename ) {
   /* uint32_t executed = 0;
    asm volatile ("\n\t"
                  "{.reg .pred p;\n\t"
                  "mov.u32 %0, 0U;\n\t"
                  "isspacep.global p, %1;\n\t"
                  "@p call __curd__glmemReadHook, (%1,%2,%3,%4);\n\t"
                  "@p mov.u32 %0, 1U;\n\t"
                  "isspacep.shared p, %1;\n\t"
                  "@p call __curd__shmemReadHook, (%1,%2,%3,%4);\n\t"
                  "@p mov.u32 %0, 2U;\n\t}"
                  : "=r"(executed) : "l"(addr), "r"((uint32_t)width), "r"(lineNo), "l"(filename) );
    printf("executed: %d\n", executed);*/
    using namespace util;
    using namespace curd;
    if( isGlobalPtr( addr ) ){
      __curd__glmemReadHook( addr, width, lineNo, filename );
    }else if( isSharedPtr( addr ) ){
      __curd__shmemReadHook( addr, width, lineNo, filename );
    } else {
      INC_COUNTER(irrelevantAccesses);
    }
  }




  __device__  __inline__  void __curd__genericMemWriteHook( uint64_t addr, uint8_t width, uint32_t lineNo, char * filename ) {
    /* uint32_t executed = 0;
    asm volatile ("\n\t"
                  "{.reg .pred %%p;\n\t"
                  "mov.u32 %0, 0U;\n\t"
                  "isspacep.global %%p, %1;\n\t"
                  "@%%p call __curd__glmemWriteHook, (%1,%2,%3,%4);\n\t"
                  "@%%p mov.u32 %0, 1U;\n\t"
                  "isspacep.shared %%p, %1;\n\t"
                  "@%%p call __curd__shmemWriteHook, (%1,%2,%3,%4);\n\t"
                  "@%%p mov.u32 %0, 2U;\n\t}"
                  : "=r"(executed) : "l"(addr), "r"((uint32_t)width), "r"(lineNo), "l"(filename) );
    printf("executed: %d\n", executed);*/
    using namespace util;
    using namespace curd;
    if( isGlobalPtr( addr ) ){
      __curd__glmemWriteHook( addr, width, lineNo, filename );
    }else if( isSharedPtr( addr ) ){
      __curd__shmemWriteHook( addr, width, lineNo, filename );
    }else {
      INC_COUNTER(irrelevantAccesses);   
    }
  }

  

  __device__ __inline__  void __curd__barrierHook2() {
 
    using namespace curd;
    using namespace util;
    #ifdef ENABLE_SAMPLE_TIME
    auto lastClock = clock64();
    auto curClock = lastClock;
    #endif
    int tid = linearizedThreadId();
    int blockId = linearizedBlockId();
    int nthreads = blockDim.x * blockDim.y * blockDim.z;
    if( linearizedBlockId() >= configParams.blocksTrackedGM ) {
      return;
    }
 #ifdef BARRIER_CHECKS   

    INC_COUNTER( barriers );
    SAMPLE_TIME(initCycles, curClock, lastClock); 
#ifdef MONITOR_SHARED_MEMORY
    if( linearizedBlockId() >= configParams.blocksTrackedSM ) {
      return;
    }
    if(sharedMemorySizePerBlock == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }

    size_t nIntraRaces = checkIntraBlockRaces( RaceKind::IntraBlockSM,
                          shmemWriteSets, shmemReadSets,
                          blockId, tid, nthreads);
    ADD_COUNTER( intraBlockRaces, nIntraRaces );
    __syncthreads();
    
    SAMPLE_TIME(SMChecksCycles, curClock, lastClock); 
    
    MAX_COUNTER( maxSMTRSSize, *shmemReadSets.getPtrToSize( blockId, tid) );
    MAX_COUNTER( maxSMTWSSize, *shmemWriteSets.getPtrToSize( blockId, tid) );
    shmemWriteSets.clearSetForThread( blockId, tid);
    shmemReadSets.clearSetForThread( blockId, tid);
#endif
#ifdef MONITOR_GLOBAL_MEMORY
    
    if(totalGlobalMemory == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }
    MAX_COUNTER( maxGMTRSSize, *glmemReadSets.getPtrToSize( blockId, tid) );
    MAX_COUNTER( maxGMTWSSize, *glmemWriteSets.getPtrToSize( blockId, tid) );

    #ifdef DEDUP_RWSETS
    auto nRemoved = glmemReadSets.dedup(blockId, tid, glmemWriteSets);
    ADD_COUNTER(entriesRemovedTRS, nRemoved);
    #endif
    auto blueBuff = _gmSizesBuffer + blockId * nthreads;
    BlockSet * _blockWriteSet = blockWriteSets.get(blockId); 
    BlockSet * _blockReadSet = blockReadSets.get(blockId);
    #ifdef EAGER_INTERBLOCK_CHECK
    //XXX: this should be ok, as the buffer has
    int oldWsSize = _blockWriteSet -> size();
    int oldRsSize = _blockReadSet-> size();
    __syncthreads();
    #endif
    int nReadSections = 0, nWriteSections = 0;
    auto _bitmap = glmemWriteSets.profileOrdering(blockId, tid, blueBuff );
   
    // the _bitmap may be null if the sets are empty
    if (_bitmap) {
      /*if (!localTid && blockId == 6) {
        _bitmap->print("xdxuxdxuxdxuxdxu", 16, 64);
        auto * _setHead = glmemWriteSets.getSetHead(blockId, 0);
        auto setSize = *glmemWriteSets.getPtrToSize(blockId, 0);
        for (auto i =  -64; i <= + 64; i++) {
          auto index = 64 + i;
          printf("setHead[%d]:(%p, %d)\n", index, _setHead[index].addr, _setHead[index].size );
        }
        for (auto i =  -64; i <= + 64; i++) {
          auto index = 512 + i;
          //printf("setHead[%d]:(%p, %d)\n", index - 1, _setHead[index - 1 ].addr, _setHead[index - 1].size );
          printf("setHead[%d]:(%p, %d)\n", index, _setHead[index].addr, _setHead[index].size );
          //printf("setHead[%d]:(%p, %d)\n", index + 1, _setHead[index + 1].addr, _setHead[index + 1].size );

        }
        _bitmap->print("xoxoxaxaxdxdxlxl", 16, 64);
        //_bitmap->print("0123456789ABCDEF", 64, 64);
        glmemWriteSets.print(blockId, 4, 64);
      } */
      auto allOrdered = glmemWriteSets.verifyOrder(blockId, tid, _bitmap, blueBuff);
      // TODO: also support !allOrdered cases 
      //CURD_ASSERT(allOrdered);
      #ifdef SYNCTHREADS_CNC
      int * redBuff = scanBuffer + static_cast<int>(blockId * configParams.initialTWSSizeGM * nthreads);
      nWriteSections = glmemWriteSets.checkAndCompressOrdered_syncthreads(blockId, tid, _bitmap, _blockWriteSet, blueBuff, redBuff);
      #else
      nWriteSections = glmemWriteSets.checkAndCompressOrdered_atomic(blockId, tid, _bitmap, _blockWriteSet, blueBuff);
      #endif
      if (!tid) {
        delete _bitmap;
      }
    }
    __syncthreads();
    _bitmap = glmemReadSets.profileOrdering(blockId, tid, blueBuff );
   
    // the _bitmap may be null if the sets are empty
    if (_bitmap) {
      auto allOrdered = glmemReadSets.verifyOrder(blockId, tid, _bitmap, blueBuff);
      // TODO: also support !allOrdered cases 
      //CURD_ASSERT(allOrdered);
      #ifdef SYNCTHREADS_CNC
      int * redBuff = scanBuffer + static_cast<int>(blockId * configParams.initialTWSSizeGM * nthreads);
      nReadSections = glmemReadSets.checkAndCompressOrdered_syncthreads<false>(blockId, tid, _bitmap, _blockReadSet, blueBuff, redBuff);
      #else
      nReadSections = glmemReadSets.checkAndCompressOrdered_atomic(blockId, tid, _bitmap, _blockReadSet, blueBuff);
      #endif

      // RW checks
      if (nWriteSections > 0 && nReadSections > 0) {
        auto * _brsBuffer = _blockReadSet -> getPtrToBufferEntry(0);
        auto * _bwsBuffer = _blockWriteSet -> getPtrToBufferEntry(0);
        // NB: assuming the bws is ordered, as it has checked for WW races
        auto iterations = ceil_div(nReadSections, nthreads);
        for (int i = 0; i < iterations; i++) {
          auto j = i * nthreads + tid;
          if (j >= nReadSections) {
            break;
          }
          auto & s = _brsBuffer[j];
          auto _s2 = binary_search<GlobalMemoryRegion, overlapping<GlobalMemoryRegion>>(
            _bwsBuffer, nWriteSections, 1, s);
          if (_s2) {
            auto si = _s2 -> intersect(s);
            //printf("[TODO]: RW race detected! on <%p,%d> \n", si.addr, si.size );
            SourceLocation c1, c2;
            auto writer = glmemWriteSets.findCulprit(blockId, si, c1);
            auto reader = glmemReadSets.findCulprit(blockId, si, c2, writer);
            if (writer >= 0 && reader >= 0) {
              reportRace(1, RaceKind::IntraBlockGM, RaceType::RW, reader, writer, c1, c2, si);
            }
          }
        }  
        
      }
      __syncthreads();
      if (!tid) {
        delete _bitmap;
      }
    }
    __syncthreads();
    // commit the merged sections 
    if (nReadSections > 0) {
      blockReadSets.updateSize(blockId, tid, nReadSections, blockReadSetsLocksDir);
    }
    if (nWriteSections > 0) {
      blockWriteSets.updateSize(blockId, tid, nWriteSections, blockWriteSetsLocksDir);
    } 
    if (!tid) {
      auto newBwsSize = blockWriteSets.get(blockId) -> size();
      auto newBrsSize = blockReadSets.get(blockId) -> size();
      MAX_COUNTER(maxReadSections, nReadSections);
      MAX_COUNTER(maxWriteSections, nWriteSections);
      MAX_COUNTER( maxBWSSize, newBwsSize );
      MAX_COUNTER( maxBRSSize, newBrsSize );
    }
    
    __syncthreads();
  
    #ifdef EAGER_INTERBLOCK_CHECK
    eagerInterBlockCheck(oldWsSize, oldRsSize, nWriteSections, nReadSections);
    __syncthreads();
    #endif
    glmemWriteSets.clearSetForThread( blockId, tid);
    glmemReadSets.clearSetForThread( blockId, tid);

#endif // end MONITOR_GLOBAL_MEMORY
#endif // end BARRIER_CHECKS
    

  }
  
    

  __device__ __inline__  void __curd__barrierHook() {
    __curd__barrierHook2();
    /*
    using namespace curd;
    using namespace util;
    #ifdef ENABLE_SAMPLE_TIME
    auto lastClock = clock64();
    auto curClock = lastClock;
    #endif
    int localTid = linearizedThreadId();
    int blockId = linearizedBlockId();
    int nthreads = blockDim.x * blockDim.y * blockDim.z;
    INC_COUNTER( barriers );
    SAMPLE_TIME(initCycles, curClock, lastClock);
    #ifdef MONITOR_SHARED_MEMORY
    if( linearizedBlockId() >= configParams.blocksTrackedSM ) {
      return;
    }
    if(sharedMemorySizePerBlock == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }

    size_t nIntraRaces = checkIntraBlockRaces( RaceKind::IntraBlockSM,
                          shmemWriteSets, shmemReadSets,
                          blockId, localTid, nthreads);
    ADD_COUNTER( intraBlockRaces, nIntraRaces );
    __syncthreads();
    
    SAMPLE_TIME(SMChecksCycles, curClock, lastClock); 
    
    MAX_COUNTER( maxSMTRSSize, *shmemReadSets.getPtrToSize( blockId, localTid) );
    MAX_COUNTER( maxSMTWSSize, *shmemWriteSets.getPtrToSize( blockId, localTid) );
    shmemWriteSets.clearSetForThread( blockId, localTid);
    shmemReadSets.clearSetForThread( blockId, localTid);
#endif
#ifdef MONITOR_GLOBAL_MEMORY
    if( linearizedBlockId() >= configParams.blocksTrackedGM ) {
      return;
    }
    if(totalGlobalMemory == 0 ) {
      printf("[ warning ] curd not initialized yet.\n");
      return ;
    }
    
    #ifdef DEDUP_RWSETS
    auto nRemoved = glmemReadSets.dedup(blockId, localTid, glmemWriteSets);
    ADD_COUNTER(entriesRemovedTRS, nRemoved);
    #endif
    SAMPLE_TIME(dedupCycles, curClock, lastClock); 
    #ifdef ENABLE_BLOCKSETS
   
    int deltaSizeWSet = 0,  deltaSizeRSet = 0;
    deltaSizeWSet = glmemWriteSets.compressIntoBlockSet_par2( blockId, localTid,
                        nthreads, blockWriteSets, _gmSizesBuffer+ blockId * nthreads);
    SAMPLE_TIME(BWSCompressCycles, curClock, lastClock); 
    //__syncthreads();
    deltaSizeRSet = glmemReadSets.compressIntoBlockSet_par2( blockId, localTid,
                        nthreads, blockReadSets, _gmSizesBuffer + blockId * nthreads);
    SAMPLE_TIME(BRSCompressCycles, curClock, lastClock); 

    auto trsSize = *glmemReadSets.getPtrToSize( blockId, localTid);
    auto twsSize = *glmemWriteSets.getPtrToSize( blockId, localTid);
    MAX_COUNTER( maxGMTRSSize, trsSize );
    MAX_COUNTER( maxGMTWSSize, twsSize );
    ADD_COUNTER( brsWorstCount, trsSize );
    ADD_COUNTER( bwsWorstCount, twsSize );

    #ifdef EAGER_INTERBLOCK_CHECK
    //XXX: this should be ok, as the buffer has
    int oldWsSize = blockWriteSets.get(blockId) -> size();
    int oldRsSize = blockReadSets.get(blockId) -> size();
    __syncthreads();
    #endif
    #ifdef INTRABLOCK_GM_CHECKS
    checkIntraBlockRaces( RaceKind::IntraBlockGM,
      glmemWriteSets, glmemReadSets,
      blockId, localTid, nthreads);
    #endif
    SAMPLE_TIME( GMIntraBChecksCycles, curClock, lastClock); 
    // the actual deltaSizes may be shrinked after committing the buffer
    deltaSizeWSet = blockWriteSets.commitBuffer( blockId, localTid,
                    deltaSizeWSet, blockWriteSetsLocksDir);
    deltaSizeRSet = blockReadSets.commitBuffer( blockId, localTid,
                    deltaSizeRSet, blockWriteSetsLocksDir );
    SAMPLE_TIME( GMCommitBufferCycles, curClock, lastClock); 
    if( !localTid ) {
      ADD_COUNTER( brsActualCount, deltaSizeRSet );
      ADD_COUNTER( bwsActualCount, deltaSizeWSet );
    }
    auto newBwsSize = blockWriteSets.get(blockId) -> size();
    auto newBrsSize = blockReadSets.get(blockId) -> size();
    MAX_COUNTER( maxBWSSize, newBwsSize );
    MAX_COUNTER( maxBRSSize, newBrsSize );
    __syncthreads();

    #ifdef EAGER_INTERBLOCK_CHECK
    eagerInterBlockCheck( oldWsSize, oldRsSize, deltaSizeWSet, deltaSizeRSet );
    __syncthreads();
    blockWriteSets.get( blockId ) -> syncSize();
    blockReadSets.get( blockId ) -> syncSize();
    // merge the thread CapabilitySets into the block CS
    //mergeThreadCapabilitySets( blockId );
    #endif

#endif

    glmemWriteSets.clearSetForThread( blockId, localTid);
    glmemReadSets.clearSetForThread( blockId, localTid);
  

#endif*/
  }
  //TODO: differentiate releases from acquires
  __device__ void __curd__atomicHook( uint64_t addr ){
    using namespace curd;
    using namespace util;
    INC_COUNTER( atomicCalls );
    
    auto blockId = linearizedBlockId();
    auto _acs = getAtomicCapabilitySet( addr );
    if( blockId >= configParams.blocksTrackedGM ){
      return;
    }
    auto lTid = linearizedBlockId();
    auto gTid = globalTid();
    // acquire
    // the capabilities gained from addr is only visible to the current 
    // thread 
    auto _tcs = getThreadCapabilitySet<true>( gTid );
    
    auto _bws = blockWriteSets.get( blockId );
    auto _brs = blockReadSets.get( blockId );
    //TODO: this is also necessary if acq & rel are separate 
    auto wsSize = *glmemWriteSets.getPtrToSize( blockId, lTid );
    auto wSfrId = -1;
    auto rSfrId = -1;
    if( wsSize ){
      wSfrId = _bws -> commitSFR( glmemWriteSets, lTid, _tcs );
    }
    auto rsSize = *glmemReadSets.getPtrToSize( blockId, lTid );
    if( rsSize ){
      rSfrId = _brs -> commitSFR( glmemReadSets, lTid, _tcs );
    }
    _tcs -> addAll( _acs ); 

    // release 
    // propagate the capabilities this block got to others
    // TODO: avoid redundant insertions
    _acs -> addAll( _tcs );

    auto wSize = _bws -> getLastSyncedSize();
    auto rSize = _brs -> getLastSyncedSize();
    if( wSize || rSize ){ // capability of the current block
      Capability c( Capability::Mode::BC, blockId, rSize, wSize  );
      _acs -> add( c );
    }
    //TODO: this separate capability may be avoided if the subsets the sfrs
    // describes are adjacent to the last synced prefix of the block set
    if( wSfrId != -1 || rSfrId != -1 ){
      Capability c( Capability::Mode::TC, blockId, rSfrId, wSfrId );
      _acs -> add( c );
    }
    

  }
}// end extern "C"

namespace curd{

  __global__ static void __curd__finalBarrierHookCall() {
    __curd__barrierHook2();
  }

#ifndef EAGER_INTERBLOCK_CHECK
  __global__ static void __curd__finalInterBlockCheck() {
    #ifdef INTERBLOCK_CHECKS
    int blockId = linearizedBlockId();
    if( blockId < configParams.blocksTrackedGM ) {
      int bwsSize = (int)blockWriteSets.get(blockId) -> size();
      int brsSize = (int)blockReadSets.get(blockId) -> size();

      int localTid= linearizedThreadId();
      int nthreads = blockDim.x * blockDim.y * blockDim.z;
      size_t WWChecks = 0;
      // XXX: we remove redundant checks at the block-level;
      // may also try to do it on the thread-level: i.e., all
      // blocks active, with each block doing half of the job
      if( blockId < ceil_div( configParams.blocksTrackedGM, 2 ) ) {
        WWChecks = blockWriteSets.checkInterBlockOverlaps(RaceType::WW, blockId, localTid,  nthreads);
        //printf("b%dt%d : WWChecks = %d, while brsSize = %d, bwsSize = %d\n", blockId, localTid, (int)WWChecks, brsSize, bwsSize);
      }
      //size_t RWChecks = blockReadSets.checkInterBlockOverlaps(RaceType::RW, blockId, localTid, nthreads, blockWriteSets);
      size_t RWChecks = blockWriteSets.checkInterBlockOverlaps(RaceType::WR, blockId, localTid, nthreads, blockReadSets);

      ADD_COUNTER( interGWWRaces, WWChecks  );
      ADD_COUNTER( interGRWRaces, RWChecks  );
      
    }
    #endif
  }
#endif


  static inline void initMetadataForShmem( size_t threadsNumPerBlock,
      size_t blocksNum, size_t smTRSCapacity, size_t smTWSCapacity) {
    using namespace std;
#ifdef MONITOR_SHARED_MEMORY
#ifdef VERBOSE_OUTPUT
    /*cout << "Allocating sets for SHMEM: " << threadsNumPerBlock <<
         " threads * " << blocksNum << " blocks * " << smTRSCapacity
         << "(R), " << smTWSCapacity << "(W)" << endl;*/
#endif
    INIT_MAPPED_COALESCED_SETS( CoalescedSetsShmem, shmemReadSets,  threadsNumPerBlock, blocksNum, smTRSCapacity);
    INIT_MAPPED_COALESCED_SETS( CoalescedSetsShmem, shmemWriteSets, threadsNumPerBlock, blocksNum, smTWSCapacity);

#endif

  }

  static inline void initMetadataForGlmem( size_t threadsNumPerBlock,
      size_t blocksNum, size_t gmTRSCapacity, size_t gmTWSCapacity,
      size_t brsCapacity, size_t bwsCapacity) {
    using namespace util;
#ifdef MONITOR_GLOBAL_MEMORY
    INIT_MAPPED_COALESCED_SETS( CoalescedSetsGlmem, glmemReadSets,  threadsNumPerBlock, blocksNum, gmTRSCapacity);
    INIT_MAPPED_COALESCED_SETS( CoalescedSetsGlmem, glmemWriteSets, threadsNumPerBlock, blocksNum, gmTWSCapacity);
    INIT_MAPPED_GRANDCENTRAL_SETS( BlockSets, blockReadSets, blocksNum, brsCapacity);
    INIT_MAPPED_GRANDCENTRAL_SETS( BlockSets, blockWriteSets, blocksNum, bwsCapacity);

    INIT_DIRECTORY( LocksDirectory, blockReadSetsLocksDir, blocksNum, blocksNum );
    INIT_DIRECTORY( LocksDirectory, blockWriteSetsLocksDir, blocksNum, blocksNum );
    INIT_DIRECTORY( VectorClock2D, blockVectorClocksDirectory, blocksNum, blocksNum );

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( blockReadSetsLocksDir),
                                         _host__blockReadSetsLocksDir, sizeof(LocksDirectory)));
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( blockWriteSetsLocksDir),
                                         _host__blockWriteSetsLocksDir, sizeof(LocksDirectory)));
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( blockVectorClocksDirectory),
                                         _host__blockVectorClocksDirectory, sizeof(VectorClock2D)));

    // allocate the global metadata map for atomics
    d_atomicCSMap = static_cast<AtomicCSMap *>( AtomicCSMap::makeDeviceHashMap( 4 ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( _atomicCSMap), &d_atomicCSMap,
      sizeof( decltype( d_atomicCSMap) ) ) );
    // allocate the cs map for blocks
    d_blockCSMap = static_cast<BlockCSMap*>( BlockCSMap::makeDeviceHashMap( 4 ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( _blockCSMap), &d_blockCSMap,
      sizeof( decltype( d_blockCSMap) ) ) );
    // allocate the cs map for threads
    d_threadCSMap = static_cast<ThreadCSMap*>( ThreadCSMap::makeDeviceHashMap( 4 ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( _threadCSMap), &d_threadCSMap,
      sizeof( decltype( d_threadCSMap) ) ) );
    // allocate the tspb map for threads
    d_tspbMap = static_cast<TSPBMap*>( TSPBMap::makeDeviceHashMap( 4 ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( _tspbMap), &d_tspbMap,
      sizeof( decltype( d_tspbMap) ) ) );




#endif

  }
  static inline hipDeviceProp_t printDeviceInfo() {
    using namespace std;
    int device;
    hipDeviceProp_t deviceProperties;
    checkCudaErrors( hipGetDevice(&device) );
    checkCudaErrors( hipGetDeviceProperties(&deviceProperties, device) );
    size_t freeDeviceMem = 0;
    checkCudaErrors( hipMemGetInfo( &freeDeviceMem, nullptr ) );
    #ifdef VERBOSE_OUTPUT
    cout << curd_msg( "NAME:" ) << deviceProperties.name << endl;
    cout << curd_msg( "SHMEM:") << deviceProperties.sharedMemPerBlock << endl;
    cout << curd_msg( "GLMEM:") << deviceProperties.totalGlobalMem <<
         ",free: " << freeDeviceMem << endl;
    cout << curd_msg( "MULTIPROCESSORS:") <<
         deviceProperties.multiProcessorCount << endl;
    cout << curd_msg( "MAX RESIDENT BLOCKS/ SM:") <<
         MAX_RESIDENT_BLOCKS_PER_SM << endl;
    #endif
    return deviceProperties;
  }
  static inline size_t computeMetaMemorySize( size_t nthreads ){
    using namespace std;
    size_t bytesOfSetBodies = sizeof( SharedMemoryRegion ) *
      (_host__configParams -> initialTRSSizeSM + _host__configParams -> initialTWSSizeSM )
         * nthreads * _host__configParams -> blocksTrackedSM;
    size_t bytesOfSizesArray = sizeof(SharedMemorySizeType) * nthreads * _host__configParams -> blocksTrackedSM;
    bytesOfSetBodies += sizeof( GlobalMemoryRegion) *
      (_host__configParams -> initialTRSSizeGM + _host__configParams -> initialTWSSizeGM )
         * nthreads * _host__configParams -> blocksTrackedGM;
    bytesOfSizesArray += sizeof(GlobalMemorySizeType) * nthreads * _host__configParams -> blocksTrackedGM;
    #ifdef VERBOSE_OUTPUT
    cout << curd_msg(" Counter ShadowMemorySize = ") << bytesOfSetBodies + bytesOfSizesArray << endl;
    #endif
    return bytesOfSetBodies + bytesOfSizesArray;
  }

  

  static inline void freeMetadataForShmem() {
#ifdef MONITOR_SHARED_MEMORY
    using namespace std;
    cout << "Freeing shmemReadSets" << endl;
    FREE_MAPPED_MEMORY( shmemReadSets );
    cout << "Freeing shmemWriteSets" << endl;
    FREE_MAPPED_MEMORY( shmemWriteSets );

#endif

  }
  __global__ void __curd__cleanUp(){
    _atomicCSMap -> clear();
    _blockCSMap -> clear();
    _threadCSMap -> clear();
    _tspbMap -> clear();
  }

  static inline void freeMetadataForGlmem() {
    using namespace std;
#ifdef MONITOR_GLOBAL_MEMORY
    cout << "Freeing glmemReadSets" << endl;

    FREE_MAPPED_MEMORY( glmemReadSets );
    cout << "Freeing glmemWriteSets" << endl;

    FREE_MAPPED_MEMORY( glmemWriteSets );
    FREE_MAPPED_MEMORY( blockReadSets );
    FREE_MAPPED_MEMORY( blockWriteSets );

    FREE_DIRECTORY( blockReadSetsLocksDir );
    FREE_DIRECTORY( blockWriteSetsLocksDir );
    FREE_DIRECTORY( blockVectorClocksDirectory );
#endif
    FREE_MAPPED_MEMORY( racesDetected );
    delete _host__configParams;

    __curd__cleanUp<<<1,1>>>();
    checkCudaErrors( hipDeviceSynchronize() );
    AtomicCSMap::destroyDeviceHashMap( d_atomicCSMap );
    BlockCSMap::destroyDeviceHashMap( d_blockCSMap );
    ThreadCSMap::destroyDeviceHashMap( d_threadCSMap );
    TSPBMap::destroyDeviceHashMap( d_tspbMap );
    #ifdef VERBOSE_OUTPUT
    std::cout << "[CURD] exiting." << std::endl;
    #endif

  }

  __global__ static void __curd__printSummary() {
    using namespace curd;
    using namespace util;

    if( linearizedThreadId() == 0 ) {
      printf("[CURD] Summary:\n");

      __threadfence();
#ifdef ENABLE_COUNTERS
      shmemRds.print();
      shmemWrs.print();
      glmemRds.print();
      glmemWrs.print();
      barriers.print();
      interGWWRaces.print();
      interGRWRaces.print();
      interGWRRaces.print();
      smTSMerges.print();
      gmTSMerges.print();
      brsWorstCount.print();
      bwsWorstCount.print();
      brsActualCount.print();
      bwsActualCount.print();
      maxSMTRSSize.print();
      maxGMTRSSize.print();
      maxBRSSize.print();
      maxSMTWSSize.print();
      maxGMTWSSize.print();
      maxBWSSize.print();
      intraBlockRaces.print();
<<<<<<< HEAD
      entriesRemovedTRS.print();  
      initCycles.print();
      SMChecksCycles.print();
      dedupCycles.print();
      BRSCompressCycles.print();
      BWSCompressCycles.print();
      GMIntraBChecksCycles.print();
      GMCommitBufferCycles.print(); 
      sortTWSCycles.print();
      sortTRSCycles.print();
      intraBWWChecksCycles.print();
      intraBRWChecksCycles.print();
      ascendingInTWSS.print();
      descendingInTWSS.print();
      maxReadSections.print();
      maxWriteSections.print();
      ascendingInTS.print();
      descendingInTS.print();
      allInOrder.print();
      allChecks.print();
      irrelevantAccesses.print();
      printf("#races: %lu\n", racesDetected.size() );
#endif
    }
    __syncthreads();
  }
  
  static void doFinalCheck(dim3 grid, dim3 threadDim ) {
    // FIXME: bug: barriers in different kernels are independent...
    __curd__finalBarrierHookCall <<<grid, threadDim>>>();
#ifdef MONITOR_GLOBAL_MEMORY
    checkCudaErrors( hipDeviceSynchronize());
#ifndef EAGER_INTERBLOCK_CHECK
    __curd__finalInterBlockCheck<<<grid, threadDim>>>();
#endif
#endif

  }

  static void resolveSourceInfo( SourceLocation & sLoc, uint32_t fnameSize, 
    std::map< const char *, const char *> & resolved  ){
    const char * fn = sLoc.filename;
    if ( resolved.count( fn ) ){
      sLoc.filename = resolved[ fn ];
      //std::cout << "resolved: "<< sLoc.filename << std::endl;
    }else if( sLoc.filename ){
      char * fname = new char[ fnameSize + 1 ];
      //std::cout << (void *)sLoc.filename << ", " << fnameSize + 1 << std::endl;
      checkCudaErrors( hipMemcpy( fname, sLoc.filename, fnameSize + 1, hipMemcpyDeviceToHost ) );
      resolved[ sLoc.filename] = fname ;  
      sLoc.filename = fname;
    }// if sLoc.filename is nullptr, do nothing
  }
  static void printRaces(){
    // print the races detected
    checkCudaErrors( hipMemcpyFromSymbol( _host__racesDetected,
                                           HIP_SYMBOL(racesDetected), sizeof( decltype( racesDetected ))));
    auto _races = _host__racesDetected -> moveToHost();
    // resolve the filename field for SourceLocation objects
    // the filename field stores a device char *, so we need to 
    // copy the actual string from device to host, and then update 
    // filename to be a host char * 
    std::map< const char *, const char * > resolved ;
    for( auto i = 0; i < _races -> hSize(); i++ ){ 
      auto & r = _races -> at( i );
      resolveSourceInfo( r.srcLoc1, r.fnameSize1, resolved );
      resolveSourceInfo( r.srcLoc2, r.fnameSize2, resolved );
    }
    _races -> printHistogram();
    for( auto p : resolved ){
      delete[] p.second;
    }
    _races -> freeH();
    delete _races;
  }
  __global__ void __curd__resetMetadata() {
    using namespace util;
    auto tid = linearizedThreadId();
    auto blockId = linearizedBlockId();
    auto blocks = configParams.blocksTrackedGM;
    if (blockId >= blocks) {
      return;
    } 
    if (!tid) {
      #ifdef MONITOR_GLOBAL_MEMORY
      auto _blockSet = blockWriteSets.get(blockId);
      _blockSet -> reset();
      _blockSet = blockReadSets.get(blockId);
      _blockSet -> reset();
      #endif
      // reset the races buffer
      if (!blockId) {
        racesDetected.reset();
      }
    }
    #ifdef MONITOR_GLOBAL_MEMORY
    #ifdef EAGER_INTERBLOCK_CHECK
    int nthreads = blockDim.x * blockDim.y * blockDim.z;
    auto iterations = ceil_div(blocks, nthreads);
    // reset vector clocks for the eager scheme
    for (int i = 0; i < iterations; i++) {
      auto remoteBid = i * nthreads + tid;
      if (remoteBid < blocks) {
        Timestamp * _syncTime = blockVectorClocksDirectory.get(blockId, remoteBid);
        *_syncTime = 0;
      }
    }
    #endif
    glmemWriteSets.clearSetForThread( blockId, tid);
    glmemReadSets.clearSetForThread( blockId, tid);
    #endif

    #ifdef MONITOR_SHARED_MEMORY
    // TODO: these should be unnecessary, as each call to the barrier hook should have cleared them 
    shmemWriteSets.clearSetForThread( blockId, tid);
    shmemReadSets.clearSetForThread( blockId, tid);
    #endif
  }
}// end namespace curd


extern "C"{
  void allocateReadWriteSets(dim3 grid, dim3 threadDim) {
    using namespace std;
    using namespace util;
    using namespace curd;
    //assert(false && "WTF");
    size_t blocksNum  = grid.x * grid.y * grid.z;
    size_t threadsNumPerBlock = threadDim.x * threadDim.y * threadDim.z;

    cout << curd_msg("Checking last config:") << lastNThreads <<", " << threadsNumPerBlock << endl;
    if (lastNThreads == threadsNumPerBlock) { // reuse 
      cout << curd_msg("Reusing existing metadata") << endl;
      return;
    } else if (lastNThreads) { // free metadata allocated last time
      cout << curd_msg("Freeing old metadata:")  << endl;
      freeMetadataForShmem();
      freeMetadataForGlmem();
      checkCudaErrors( hipFree( _host_gmSizesBuffer ) );
      checkCudaErrors(hipFree(_host__scanBuffer));
      checkCudaErrors(hipFree(_host__bitmapBuffer));
    }
    lastNThreads = threadsNumPerBlock;
    //CURD_ASSERT(blocksNum <= MAX_BLOCKS_NUM);
    //CURD_ASSERT(threadsNumPerBlock <= MAX_THREADS_NUM);

    _host__configParams = new Config();
    _host__configParams->load( blocksNum);

    hipDeviceProp_t deviceProperties = printDeviceInfo();

    #ifdef VERBOSE_OUTPUT
    cout << curd_msg( "MODE:" ) << CURD_MODE << endl;
    cout << curd_msg( "BLOCKS:") << blocksNum << endl;
    cout << curd_msg( "THREADS:") << threadsNumPerBlock << endl;
    #endif
    checkCudaErrors( hipDeviceSynchronize());

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sharedMemorySizePerBlock),
                                        &(deviceProperties.sharedMemPerBlock), sizeof(size_t)));
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(totalGlobalMemory),
                                        &(deviceProperties.totalGlobalMem), sizeof(decltype(totalGlobalMemory ))));


    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( configParams), _host__configParams,
      sizeof( decltype(configParams)) ));
    // allocate buffer for recording races
    _host__racesDetected = new RaceBuffer( GMEM_RACES_BUFFER_SIZE, 0 );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( racesDetected),_host__racesDetected,
      sizeof(decltype(racesDetected ) )));

    initMetadataForShmem( threadsNumPerBlock, _host__configParams->blocksTrackedSM,
                          _host__configParams->initialTRSSizeSM, _host__configParams->initialTWSSizeSM );
    initMetadataForGlmem( threadsNumPerBlock, _host__configParams->blocksTrackedGM,
                          _host__configParams->initialTRSSizeGM, _host__configParams->initialTWSSizeGM,
                          _host__configParams->initialBRSSize, _host__configParams->initialBWSSize );
    
    computeMetaMemorySize( threadsNumPerBlock );
    // allocate block-wide size buffer 
    checkCudaErrors( hipMalloc( &_host_gmSizesBuffer, 
      sizeof(decltype(*_host_gmSizesBuffer))* threadsNumPerBlock 
        * _host__configParams-> blocksTrackedGM ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( _gmSizesBuffer), &_host_gmSizesBuffer,
      sizeof(decltype(_host_gmSizesBuffer)) ) );

    checkCudaErrors( hipMalloc( &_host__scanBuffer, 
      sizeof(decltype(*_host__scanBuffer))* threadsNumPerBlock 
        * _host__configParams-> initialTWSSizeGM
        * _host__configParams-> blocksTrackedGM ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( scanBuffer), &_host__scanBuffer,
      sizeof(decltype(_host__scanBuffer)) ) );

    checkCudaErrors( hipMalloc( &_host__bitmapBuffer, 
      sizeof(decltype(*_host__bitmapBuffer))* threadsNumPerBlock 
        * _host__configParams-> initialTWSSizeGM
        * _host__configParams-> blocksTrackedGM ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( bitmapBuffer), &_host__bitmapBuffer,
      sizeof(decltype(_host__bitmapBuffer)) ) );
    checkCudaErrors(hipDeviceSynchronize());

  }
  void freeReadWriteSets(dim3 grid, dim3 threadDim) {
    using namespace util;
    using namespace curd;

    checkCudaErrors( hipDeviceSynchronize());
    doFinalCheck(grid, threadDim);
    checkCudaErrors( hipDeviceSynchronize());
    #ifdef PRINT_SUMMARY
    __curd__printSummary<<<1,1>>>();
    #endif 
    checkCudaErrors( hipDeviceSynchronize());
    
    printRaces();
    __curd__resetMetadata<<<grid, threadDim>>>();
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef DIE_AFTER_SINGLE_RUN
    exit(0);
#endif

  }

}// end extern "C"




