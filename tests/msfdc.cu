#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#define SIZE 16
__global__ void msfdc( int * g_data){
  auto id = threadIdx.x + blockIdx.x * blockDim.x;
  g_data[ id % SIZE] += id;
  __syncthreads();
}

int main(int argc, char ** argv){
  using namespace std;
  int blocks = 2, threads = 32;
  if( argc >= 3 ){
    blocks = atoi( argv[1] );
    threads = atoi( argv[2] );
  }
  cout << " blocks = " << blocks << endl;
  cout << " threads = " << threads << endl;
  int * d_data = nullptr;
  checkCudaErrors( hipMalloc( &d_data, sizeof(int) * SIZE ));
  msfdc<<< blocks, threads >>>( d_data);
  checkCudaErrors( hipDeviceSynchronize() );

}
