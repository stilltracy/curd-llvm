#include "hip/hip_runtime.h"
#include "sort.cuh"
#include <cstdio>

template<typename T>
__host__ __device__ void print(T * array, size_t n, const char * prefix, const char * suffix  ){
  printf("%s[", prefix);
  for( int i = 0; i < n ; i ++ ){
    printf("%d ", array[i]);
  }
  printf("]%s", suffix);
}

template<typename T>
struct eq{
  __host__ __device__ bool operator()(const T & lhs, const T & rhs){
    return lhs == rhs ;
  }
};
#define size 12
__global__ void test(){
  int stride = 1;
  int a[size] = {3,2,6,1,7,9,12,4,5,8,11,10};
  
  for( int i = 0; i < stride; i++ ){
    //curd::bubble_sort(a + i, 12, 4 );
    //curd::quick_sort_iterative(a + i, stride, 0, size/stride -1 );
    //curd::merge_sort_iterative(a + i, size/stride , stride ); 
    curd::radix_sort<int,int>(a + i, size, stride );
    print(a, 12, "after sort:", "\n"); 
  }
  
  for( int i = -1; i < 14; i ++ ){
    for( int s = 1; s <=4; s ++ ){
      int * _r = curd::binary_search<int, eq<int>>(&a[0], 12/s, s , i ); 
      printf("%d (stride %d): %x\n", i, s, _r);
    }
  }
}

int main(){
  test<<<1,1>>>();
  hipDeviceSynchronize();
}
