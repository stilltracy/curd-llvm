#include "hip/hip_runtime.h"
#include "race_detection.cu"

#define BUFFER_SIZE 1024
__device__ int buffer[BUFFER_SIZE];



__global__ void testPrefixSumInclusive()
{
  int size = blockDim.x;
  int selfTid = threadIdx.x;
  GlobalMemoryRegion * regions = newArrayForBlock<GlobalMemoryRegion>(selfTid, size) ;
  GlobalMemoryRegion * mergedRegions = newArrayForBlock<GlobalMemoryRegion>(selfTid, size);
  
  GlobalMemorySizeType setSizes[8] = {4, 3, 4, 4, 1, 4, 4, 4 }; 
  GlobalMemorySizeType columnsCount = parallelMax( setSizes, size, 0, selfTid );

  int width = 4;
  regions[ selfTid] = GlobalMemoryRegion(selfTid * width, width); 
  __syncthreads();
  if(selfTid == size -1)
  {
   regions[ selfTid] = GlobalMemoryRegion((selfTid + 1 )* width, width); 
   
  }
  __syncthreads();
  assert(columnsCount == 4);
  printf("columnsCount %ld\n", columnsCount);
  for(GlobalMemorySizeType i = 0; i < columnsCount; i++)
  {
    buffer[selfTid ] = 0;
    __syncthreads();
    printArray("regions", regions, size, selfTid);
    //prefixSumExclusive( buffer, size, size, selfTid);
    int * breaks = buffer;

    bool needsToWork = i < setSizes[selfTid];
    bool isLeft = isLeftBreak(regions, columnsCount, setSizes, 
                                size, selfTid, needsToWork);
    breaks[selfTid] = isLeft;    
    __syncthreads();

    printArray("before scan", buffer, size, selfTid);
    int totalSum = prefixSumExclusive(breaks, size, size, selfTid);
    printArray("after scan", buffer, size, selfTid); 
    //post-condition: breaks saves the exclusive prefix sum of each element in the original array 
    __syncthreads();
    if( isLeft )
    {
      reduceSubRegion( regions, breaks, selfTid, size );
      // post-condition: the merged region is saved in mergedColumn[selfTid]
      int breakId = breaks[selfTid];
      mergedRegions[0 + breakId ] = regions[selfTid];
    }
    __syncthreads();
    
    int maxBreakId = breaks[ size -1 ];
    printf("i %ld, thread %d, maxBreakId %d, totalSum : %d\n", (uint64_t)i, selfTid, maxBreakId, totalSum);
    printArray("mergedRegions", mergedRegions, totalSum, selfTid);
  }
}
int main()
{
  int blocks = 1;
  int threads = 8;
  allocateReadWriteSets(blocks, threads);
  testPrefixSumInclusive<<<blocks, threads>>>();
  freeReadWriteSets(blocks, threads);
  checkCudaErrors( hipDeviceSynchronize() );
}
