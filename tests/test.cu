#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include "scalarProd_kernel.cuh"
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

const int VECTOR_N = 256;
//Number of elements per vector; arbitrary,
//but strongly preferred to be a multiple of warp size
//to meet memory coalescing constraints
const int ELEMENT_N = 4096;
//Total number of data elements
const int    DATA_N = VECTOR_N * ELEMENT_N;

const int   DATA_SZ = DATA_N * sizeof(float);
const int RESULT_SZ = VECTOR_N  * sizeof(float);

float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

__global__ void msfdc(){
  __shared__ int wtf[4];
  printf(" msfdc: %lp\n", wtf);
  __syncthreads();
}

int main(int argc, char **argv){
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
    float *d_A, *d_B, *d_C;
    double delta, ref, sum_delta, sum_ref, L1norm;
    StopWatchInterface *hTimer = NULL;
    int i;
    msfdc<<<1,1>>>();
    hipDeviceSynchronize();

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
   // findCudaDevice(argc, (const char **)argv);
    
    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory.\n");
    h_A     = (float *)malloc(DATA_SZ);
    h_B     = (float *)malloc(DATA_SZ);
    h_C_CPU = (float *)malloc(RESULT_SZ);
    h_C_GPU = (float *)malloc(RESULT_SZ);

    printf("...allocating GPU memory.\n");

	checkCudaErrors(hipMalloc((void **)&d_C, RESULT_SZ ));
	checkCudaErrors(hipMalloc((void **)&d_B, DATA_SZ ));
	checkCudaErrors(hipMalloc((void **)&d_A, DATA_SZ ));    
	printf("...generating input data in CPU mem.\n");
    srand(123);

    //Generating input data on CPU
    for (i = 0; i < DATA_N; i++)
    {
	h_A[i] = RandFloat(0.0f, 1.0f);
	h_B[i] = RandFloat(0.0f, 1.0f);
    }

    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    checkCudaErrors(hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice));
    printf("Data init done.\n");

	scalarProdGPU<<<128,256>>>( d_C, d_A, d_B, VECTOR_N , ELEMENT_N );
	checkCudaErrors(hipDeviceSynchronize() );
}
