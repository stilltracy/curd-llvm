#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#define STRIDE 4
#define SIZE 256

// 0124589
// 0123 5678 10111213
__global__ void msfdc( int * g_data, int v){
 
  auto tid =threadIdx.x + blockIdx.x * blockDim.x;
  auto id = tid % (STRIDE - 1);
  auto idx = id + (tid / (STRIDE-1) ) * STRIDE ;
  for( int i = 0; i < v ; i++ ){
    if( idx < SIZE ){
      g_data[ idx  ] += id;
      printf("b%dt%d writes g_data[%d]\n", blockIdx.x , threadIdx.x, idx );
    }
    __syncthreads();
  }
  
}

int main(int argc, char ** argv){
  using namespace std;
  int blocks = 2, threads = 32, iterations = 1;
  if( argc >= 4 ){
    blocks = atoi( argv[1] );
    threads = atoi( argv[2] );
    iterations = atoi( argv[3] );
  }
  cout << " blocks = " << blocks << endl;
  cout << " threads = " << threads << endl;
  int * d_data = nullptr;
  checkCudaErrors( hipMalloc( &d_data, sizeof(int) * SIZE ));
  msfdc<<< blocks, threads >>>( d_data, iterations);
  checkCudaErrors( hipDeviceSynchronize() );

}
